#include "hip/hip_runtime.h"
﻿#include "DirectMethod.cuh"

__host__ void calculateExponentialRVs(double* randomVariables, double* propscan, int s, int m, hipStream_t stream) {
	int num_blocks = s / MAX_THREADS_PER_BLOCK;
	int remainder = s % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		// if we have more elements than fit in a single block and the number of elements is evenly divisible by the number of threads per block, then we need only need one block per max_threads elements
		calculateExponentialRVsKernel <<<num_blocks, MAX_THREADS_PER_BLOCK, 0, stream >>> (randomVariables, propscan, s, m);
	}
	else if (num_blocks >= 1 && remainder != 0) {
		// if we have more elements than fit in a single block and the number of elements is not evenly divisible by the number of threads per block, we need an extra block to process the left overs
		calculateExponentialRVsKernel <<<num_blocks + 1, MAX_THREADS_PER_BLOCK, 0, stream >>> (randomVariables, propscan, s, m);
	}
	else {
		// Otherwise we only need one block because s < max threads so we initialize with s threads
		calculateExponentialRVsKernel <<<1, s, 0, stream >>> (randomVariables, propscan, s, m);
	}
}

// TESTER: Zhecheng
// the first half of the RV will be exponential
// the second half of the RVs will be uniform
__host__ void drawRVs(double* propscan, double* randomVariables, int s, int m, hiprandGenerator_t gen, hipStream_t stream, bool states_only) {
	if (states_only) {
		hiprandGenerateUniformDouble(gen, randomVariables + s, s);
	}
	else {
		hiprandGenerateUniformDouble(gen, randomVariables, 2 * s); // params: hiprand RNG, ptr to output, size; range: [0, 1]
		calculateExponentialRVs(randomVariables, propscan, s, m, stream);
	}
}

// TESTER: Tarek
// double[] uniformRVs: array of uniform RVs on device
// double[] propensity_scan: scanned array of propensities on device
// int s: the number of simulations AKA the number of RVs drawn
// int m: the number of reactions AKA the offset used to multiply by the final column
__host__ void scaleRVs(double* uniformRVs, double* propensity_scan, int s, int m, hipStream_t stream) {
	int num_blocks = s / MAX_THREADS_PER_BLOCK;
	int remainder = s % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		// if we have more elements than fit in a single block and the number of elements is evenly divisible by the number of threads per block, then we need only need one block per max_threads elements
		offsetMultiplicationKernel<<<num_blocks, MAX_THREADS_PER_BLOCK, 0, stream >>>(uniformRVs, propensity_scan, m, s);
	}
	else if (num_blocks >= 1 && remainder != 0) {
		// if we have more elements than fit in a single block and the number of elements is not evenly divisible by the number of threads per block, we need an extra block to process the left overs
		offsetMultiplicationKernel<<<num_blocks + 1, MAX_THREADS_PER_BLOCK, 0, stream >>>(uniformRVs, propensity_scan, m, s);
	}
	else {
		// Otherwise we only need one block because s < max threads so we initialize with s threads
		offsetMultiplicationKernel<<<1, s, 0, stream >>>(uniformRVs, propensity_scan, m, s);
	}
}

// double[] propensity_scan: scanned propensities
// double[] uniformRVs: Uniform random variables scaled to match propensities
// int s: number of simulations
// int m: number of reactions
__host__ void checkBins(double* propensity_scan, double* uniformRVs, int* bins, int s, int m, hipStream_t stream) {
	int num_blocks = m / MAX_THREADS_PER_BLOCK;
	int remainder = m % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		// All threads in all blocks will be engaged, so no need for an extra block
		dim3 gsize(num_blocks, s);
		checkBinsKernel<<<gsize, MAX_THREADS_PER_BLOCK, 0, stream >>>(propensity_scan, uniformRVs, bins, s, m);
	}
	else if (num_blocks >= 1 && remainder != 0) {
		// Last block will not have all threads engaged, this is the extra block
		dim3 gsize(num_blocks + 1, s);
		checkBinsKernel<<<gsize, MAX_THREADS_PER_BLOCK, 0, stream >>>(propensity_scan, uniformRVs, bins, s, m);
	}
	else {
		// Entire calculation can be run in one block
		dim3 gsize(1, s);
		checkBinsKernel<<<gsize, m, 0, stream >>>(propensity_scan, uniformRVs, bins, s, m);
	}
}

__host__ void updateSims(int s, int n, int* fired_reactions, int* sim_configs, int* state_changes, bool* stability_flags, hipStream_t stream) {
	int num_blocks = n / MAX_THREADS_PER_BLOCK;
	int remainder = n % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		dim3 gsize(num_blocks, s);
		updateSimsKernel << <gsize, MAX_THREADS_PER_BLOCK, 0, stream >> > (s, n, fired_reactions, sim_configs, state_changes, stability_flags);
	}
	if (num_blocks >= 1 && remainder != 0) {
		dim3 gsize(num_blocks + 1, s);
		updateSimsKernel << <gsize, MAX_THREADS_PER_BLOCK, 0, stream >> > (s, n, fired_reactions, sim_configs, state_changes, stability_flags);
	}
	else {
		dim3 gsize(1, s);
		updateSimsKernel << <gsize, n, 0, stream >> > (s, n, fired_reactions, sim_configs, state_changes, stability_flags);
	}
}

__host__ void updateProps(int s, int n, int m, int max_reactants, int* sim_configs, int* reactants, double* reaction_rates, double* propensities, hipStream_t stream) {
	int num_blocks = m / MAX_THREADS_PER_BLOCK;
	int remainder = m % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		dim3 gsize(num_blocks, s);
		updatePropsKernel << <gsize, MAX_THREADS_PER_BLOCK, 0, stream >> > (s, n, m, max_reactants, sim_configs, reactants, reaction_rates, propensities);
	}
	if (num_blocks >= 1 && remainder != 0) {
		dim3 gsize(num_blocks + 1, s);
		updatePropsKernel << <gsize, MAX_THREADS_PER_BLOCK, 0, stream >> > (s, n, m, max_reactants, sim_configs, reactants, reaction_rates, propensities);
	}
	else {
		dim3 gsize(1, s);
		updatePropsKernel << <gsize, m, 0, stream >> > (s, n, m, max_reactants, sim_configs, reactants, reaction_rates, propensities);
	}
}

__host__ void updateTimes(double* randomVariables, double* times, int s, hipStream_t stream) {
	int num_blocks = s / MAX_THREADS_PER_BLOCK;
	int remainder = s % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		// if we have more elements than fit in a single block and the number of elements is evenly divisible by the number of threads per block, then we need only need one block per max_threads elements
		updateTimesKernel << <num_blocks, MAX_THREADS_PER_BLOCK, 0, stream >> > (randomVariables, times, s);
	}
	else if (num_blocks >= 1 && remainder != 0) {
		// if we have more elements than fit in a single block and the number of elements is not evenly divisible by the number of threads per block, we need an extra block to process the left overs
		updateTimesKernel << <num_blocks + 1, MAX_THREADS_PER_BLOCK, 0, stream >> > (randomVariables, times, s);
	}
	else {
		// Otherwise we only need one block because s < max threads so we initialize with s threads
		updateTimesKernel << <1, s, 0, stream >> > (randomVariables, times, s);
	}
}	

__host__ void propCheck(double* propscan, bool* stability_flags, int s, int m, hipStream_t stream) {
	int num_blocks = s / MAX_THREADS_PER_BLOCK;
	int remainder = s % MAX_THREADS_PER_BLOCK;

	if (num_blocks >= 1 && remainder == 0) {
		propCheckKernel<<< num_blocks, MAX_THREADS_PER_BLOCK, 0, stream >> > (propscan, stability_flags, s, m);
	}
	if (num_blocks >= 1 && remainder != 0) {
		propCheckKernel<<< num_blocks + 1, MAX_THREADS_PER_BLOCK, 0, stream >> > (propscan, stability_flags, s, m);
	}
	else {
		propCheckKernel << <1, s, 0, stream>> > (propscan, stability_flags, s, m);
	}
}

__host__ thrust::host_vector<int> build_full_pci(std::vector<int> pci, int s) {
	thrust::host_vector<int> full_pci(s * pci.size());
	for (int idx = 0; idx < s; idx++) {
		thrust::copy(thrust::host, pci.begin(), pci.end(), &full_pci[idx * pci.size()]);
	}
	return full_pci;
}

// TODO: Multi-gpu support
// TODO: Make randomVariables resize when using --states-only

// Loads data into device memory, launches kernels, and runs until final iteration
// int[] state_change_matrix: flattened matrix whose rows are state change vectors corresponding to reactions by row-index and species by column-index
// double[] rrc_vector: an array of rate reaction constants corresponding to reactions by index
// int[] configuration_matrix: a flattened matrix of simulation configurations corresponding to simulations by row-index and species by column-index
// double[] propensity_matrix: a flattened matrix of propensities corresponding to simulations by row-index and reactions by column-index
// int[] reactants_table: a flattened reverse-lookup table; row-indices correspond to reactions, column-indices correspond to reactant id/count pairs; should be padded to the max number of reactants with (0, 0)
// int s: the number of simulations
// int n: the number of species
// int m: the number of reactions
// int max_reactants: the maximum number of reactants in any reaction
// int stop: the number of iterations to run
// bool verbose: Whether to print status messages to the console.
__host__ void directMethod(int* state_change_matrix, double* rrc_vector, int* configuration_matrix, double* propensity_matrix, 
	int* reactants_table, int s, int n, int m, int max_reactants, int stop, bool verbose, bool all_confs, bool stability_only, 
	bool early_stop, bool debug, bool states_only, std::string rng, unsigned long long seed, std::vector<int> pci) {

	bool partial = pci.size() > 0;
	std::string fext = ".bin";

	if (verbose) {
		std::cout << "Constructing variables and moving data to device..." << std::endl << std::flush;
	}

	// Construct keys for row-wise inclusive scan
	int* host_keys = new int[s * m];
	for (int i = 0; i < s; i++) {
		for (int j = 0; j < m; j++) {
			host_keys[i * m + j] = i; // key for each element in a row is just its row number
		}
	}

	// build full list of copy indices for --partial-copy
	thrust::host_vector<int> h_full_pci = build_full_pci(pci, s);

	// Move input data to device
	thrust::device_vector<int> scm(state_change_matrix, state_change_matrix + n * m);
	thrust::device_vector<double> rrc_vec(rrc_vector, rrc_vector + m);
	thrust::device_vector<int> confmat(configuration_matrix, configuration_matrix + s * n);
	thrust::device_vector<double> propmat(propensity_matrix, propensity_matrix + s * m);
	thrust::device_vector<int> reactants(reactants_table, reactants_table + m * max_reactants * 2);
	thrust::device_vector<int> keys(host_keys, host_keys + s * m);
	thrust::device_vector<double> times(s, 0); // all times are 0 to start
	
	// Allocate output locations on device
	thrust::device_vector<double> propscan(s * m);
	thrust::device_vector<double> randomVariables(2*s); // Contains both exponential and uniform RVs (in first and second halves respectively)
	thrust::device_vector<int> bins(s);
	thrust::device_vector<bool> stability_flags(s, false); // Each value indicates whether the simulation has reached stability (all propensities zero in that sim), assumed false at start
	thrust::device_vector<int> partial_confmat(s * pci.size());
	thrust::device_vector<int> full_pci = h_full_pci;

	// Allocate output locations on host with pinned memory
	pinnedBoolVector h_stability_flags(s, false);
	pinnedIntVector h_confmat(configuration_matrix, configuration_matrix + s * n);
	pinnedDoubleVector h_times(s, 0);
	
	// Fill partial confmat with initial partial config for record keeping
	pinnedIntVector h_partial_confmat(s * pci.size());
	if (partial) {
		thrust::gather(thrust::host, h_full_pci.begin(), h_full_pci.end(), configuration_matrix, h_partial_confmat.begin());
	}

	// Pinned memory pointers for saving binaries and early stopping
	bool* h_stability_flags_ptr = thrust::raw_pointer_cast(h_stability_flags.data());
	int* h_confmat_ptr = thrust::raw_pointer_cast(h_confmat.data());
	double* h_times_ptr = thrust::raw_pointer_cast(h_times.data());
	int* h_partial_confmat_ptr = thrust::raw_pointer_cast(h_partial_confmat.data());

	// Raw device pointers for kernel processing outside thrust
	int* scm_ptr = thrust::raw_pointer_cast(scm.data());
	int* confmat_ptr = thrust::raw_pointer_cast(confmat.data());
	int* reactants_ptr = thrust::raw_pointer_cast(reactants.data());
	int* bins_ptr = thrust::raw_pointer_cast(bins.data());
	int* partial_confmat_ptr = thrust::raw_pointer_cast(partial_confmat.data());

	double* rrc_ptr = thrust::raw_pointer_cast(rrc_vec.data());
	double* uniformRVs_ptr = thrust::raw_pointer_cast(randomVariables.data()) + s;
	double* exponentialRVs_ptr = thrust::raw_pointer_cast(randomVariables.data());
	double* propmat_ptr = thrust::raw_pointer_cast(propmat.data());
	double* propscan_ptr = thrust::raw_pointer_cast(propscan.data());
	double* times_ptr = thrust::raw_pointer_cast(times.data());

	bool* stability_flags_ptr = thrust::raw_pointer_cast(stability_flags.data());

	if (verbose) {
		std::cout << "Setting stream policy..." << std::endl << std::flush;
	}

	// Create streams for kernel launches and memcpyasync (default stream used if not set)
	hipStream_t kernel_stream, memcpy_stream;
	if (all_confs || stability_only || early_stop) {
		hipStreamCreate(&kernel_stream);
		hipStreamCreate(&memcpy_stream);
	}
	else {
		kernel_stream = 0; // 0 is default stream
		memcpy_stream = 0;
	}

	if (verbose) {
		std::cout << "Building random number generator..." << std::endl << std::flush;
	}

	// TODO: Fix exponential random variables
	// Make hiprand generator
	hiprandGenerator_t gen;
	hiprandRngType_t gen_type;
	
	if (rng == "XORWOW") {
		gen_type = HIPRAND_RNG_PSEUDO_XORWOW;
	}
	else if (rng == "MRG32K3A") {
		gen_type = HIPRAND_RNG_PSEUDO_MRG32K3A;
	}
	else if (rng == "MTGP32") {
		gen_type = HIPRAND_RNG_PSEUDO_MTGP32;
	}
	else if (rng == "PHILOX-4X32-10") {
		gen_type = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
	}
	else if (rng == "SOBOL32") {
		gen_type = HIPRAND_RNG_QUASI_SOBOL32;
	}
	else if (rng == "SOBOL64") {
		gen_type = HIPRAND_RNG_QUASI_SOBOL64;
	}
	else if (rng == "scrambledSOBOL32") {
		gen_type = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
	}
	else if (rng == "scrambledSOBOL64") {
		gen_type = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
	}
	else {
		gen_type = HIPRAND_RNG_PSEUDO_MT19937;
	}
	
	hiprandCreateGenerator(&gen, gen_type); // create MT19937 cuRAND RNG
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);	// gen, seed

	// Make hiprand execute on kernel stream
	hiprandSetStream(gen, kernel_stream);

	// Make timestamp to ensure unique output binary filenames
	time_t rawtime;
	struct tm* timeinfo;
	char buffer[80];

	time(&rawtime);
	timeinfo = localtime(&rawtime);

	strftime(buffer, sizeof(buffer), "%d-%m-%Y_%H-%M-%S", timeinfo);
	std::string timestamp(buffer);

	// Save initial config for consistency, prior to main loop since it is not stored yet
	if (all_confs) {
		if (partial) {
			save_config("out/config0_" + timestamp + fext, std::vector<int>(h_partial_confmat.begin(), h_partial_confmat.end()));
		}
		else {
			save_config("out/config0_" + timestamp + fext, std::vector<int>(state_change_matrix, state_change_matrix + s * n));
		}

		if (!states_only) {
			save_times("out/times0_" + timestamp + fext, std::vector<double>(s * n, 0));
		}
	}

	// Only used if debug is set
	thrust::host_vector<double> h_propscan(s * m);
	thrust::host_vector<double> h_props(s * m);
	thrust::host_vector<bool> h_stabflags(s);
	thrust::host_vector<int> h_bins(s);
	thrust::host_vector<double> h_rvs(2*s);
	thrust::host_vector<double> h_chrono(s);
	thrust::host_vector<int> h_gather_out(s * pci.size());
	thrust::host_vector<int> h_configmat(s * n);

	// if not debugging deallocate to save space.
	if (!debug) {
		h_propscan.clear();
		h_propscan.shrink_to_fit();

		h_props.clear();
		h_props.shrink_to_fit();

		h_stabflags.clear();
		h_stabflags.shrink_to_fit();

		h_bins.clear();
		h_bins.shrink_to_fit();

		h_rvs.clear();
		h_rvs.shrink_to_fit();

		h_chrono.clear();
		h_chrono.shrink_to_fit();
	}

	int idx = 0;
	while(idx < stop || stability_only) {
		if (verbose && !debug) {
			if (stability_only) {
				std::cout << "\rRunning iteration [" << idx + 1 << "]" << std::flush;
			}
			else {
				std::cout << "\rRunning iteration [" << idx + 1 << "/" << stop << "]" << std::flush;
			}
		}

		// Update propensities
		updateProps(s, n, m, max_reactants, confmat_ptr, reactants_ptr, rrc_ptr, propmat_ptr, kernel_stream);

		// Inclusive scan of each row
		thrust::inclusive_scan_by_key(thrust::cuda::par.on(kernel_stream), keys.begin(), keys.end(), propmat.begin(), propscan.begin());

		// Set stability flags
		propCheck(propscan_ptr, stability_flags_ptr, s, m, kernel_stream);

		// Draw RVs
		drawRVs(propscan_ptr, exponentialRVs_ptr, s, m, gen, kernel_stream, states_only);

		// Update times
		updateTimes(exponentialRVs_ptr, times_ptr, s, kernel_stream); //TODO: stop time update once stability has been reached

		// Scale uniform RVs with element-wise multiplication
		scaleRVs(uniformRVs_ptr, propscan_ptr, s, m, kernel_stream);

		// Checkbins kernel
		checkBins(propscan_ptr, uniformRVs_ptr, bins_ptr, s, m, kernel_stream);

		if (debug) {
			// test output
			h_propscan = propscan;
			h_props = propmat;
			h_stabflags = stability_flags;
			h_bins = bins;
			h_rvs = randomVariables;
			h_chrono = times;
			h_gather_out = h_partial_confmat;
			h_configmat = confmat;
		}

		// Update state
		updateSims (s, n, bins_ptr, confmat_ptr, scm_ptr, stability_flags_ptr, kernel_stream);

		// Gather desired counts into temporary buffer
		if (partial) {
			thrust::gather(thrust::cuda::par.on(kernel_stream), full_pci.begin(), full_pci.end(), confmat.begin(), partial_confmat.begin());
		}

		// Let kernels get loaded before running blocking host code

		// Synchronize with device so that memory transfers from last round can finish executing.
		hipDeviceSynchronize();

		if (debug) {
			std::cout << std::endl << std::endl;
			std::cout << "Iteration " << idx << std::endl;

			std::cout << "Stability Flags = {";
			for (int i = 0; i < s; i++) {
				std::cout << h_stabflags[i] << ", ";
			}
			std::cout << "}" << std::endl << std::endl;

			std::cout << "Configuration Matrix = {";
			for (int i = 0; i < s; i++) {
				std::cout << "{";
				for (int j = 0; j < n; j++) {
					std::cout << h_configmat[i * n + j] << ", ";
				}
				std::cout << "}, " << std::endl;
			}
			std::cout << "}" << std::endl;

			std::cout << "Exponential RVs = {";
			for (int i = 0; i < s; i++) {
				std::cout << h_rvs[i] << ", ";
			}
			std::cout << "}" << std::endl;

			std::cout << "Time Variables = {";
			for (int i = 0; i < s; i++) {
				std::cout << h_chrono[i] << ", ";
			}
			std::cout << "}" << std::endl << std::endl;

			std::cout << "Propensity Sums = {";
			for (int i = 0; i < s; i++) {
				std::cout << h_propscan[m + i * m - 1] << ", ";
			}
			std::cout << "}" << std::endl;

			std::cout << "Uniform RVs = {";
			for (int i = 0; i < s; i++) {
				std::cout << h_rvs[s + i] << ", ";
			}
			std::cout << "}" << std::endl;

			std::cout << "Propensity bounds = {";
			for (int i = 0; i < s; i++) {
				std::cout << "(" << h_propscan[i*m + h_bins[i] - 1] << ", " << h_propscan[i*m + h_bins[i]] << "), ";
			}
			std::cout << "}" << std::endl << std::endl;

			if (partial) {
				std::cout << "Partial config = {";
				for (int i = 0; i < pci.size(); i++) {
					std::cout << h_gather_out[i] << ", ";
				}
				std::cout << "}" << std::endl;
			}
		}
		
		// Don't run saving and checks on the first iteration since no device data hasn't been copied back yet
		if (idx > 0) {
			if (all_confs) {
				std::string fname = "out/config" + std::to_string(idx) + "_" + timestamp + fext;
				if (partial) {
					h_partial_confmat = partial_confmat;
					save_config(fname, std::vector<int>(h_partial_confmat_ptr, h_partial_confmat_ptr + s * pci.size()));
				}
				else {
					h_confmat = confmat;
					save_config(fname, std::vector<int>(h_confmat_ptr, h_confmat_ptr + s * n));
				}
				fname = "out/times" + std::to_string(idx) + "_" + timestamp + fext;

				if (!states_only) {
					save_times(fname, std::vector<double>(h_times_ptr, h_times_ptr + s));
				}
			}
			bool stability_reached = is_stable(h_stability_flags_ptr, s);
			if ((stability_only || early_stop) && stability_reached) {
				break;
			}
		}

		// asynchronously move data from device to host in seperate stream from kernels.
		// Should only not occur if we are running in vanilla, iterations-only, no-saving mode.
		if (all_confs || early_stop || stability_only) {
			if (partial) {
				hipMemcpyAsync(h_partial_confmat_ptr, partial_confmat_ptr, s * pci.size() * sizeof(int), hipMemcpyDeviceToHost, memcpy_stream);
			}
			else {
				hipMemcpyAsync(h_confmat_ptr, confmat_ptr, s * n * sizeof(int), hipMemcpyDeviceToHost, memcpy_stream);
			}
			hipMemcpyAsync(h_stability_flags_ptr, stability_flags_ptr, s * sizeof(bool), hipMemcpyDeviceToHost, memcpy_stream);
		}

		++idx;
	}

	if (!debug) {
		std::cout << std::endl << std::endl;
	}

	// Allow final memmory transfer to occur before continuing, if it is still running.
	hipDeviceSynchronize();

	// Transfer of final config and save it
	if (partial) {
		h_partial_confmat = partial_confmat;
		save_config("out/config" + std::to_string(idx) + "_" + timestamp + fext, std::vector<int>(h_partial_confmat_ptr, h_partial_confmat_ptr + s * pci.size()));
	}
	else {
		h_confmat = confmat;
		save_config("out/config" + std::to_string(idx) + "_" + timestamp + fext, std::vector<int>(h_confmat_ptr, h_confmat_ptr + s * n));
	}

	if (!states_only) {
		h_times = times;
		save_times("out/times" + std::to_string(idx) + "_" + timestamp + fext, std::vector<double>(h_times_ptr, h_times_ptr + s));
	}

	if (verbose) {
		std::cout << "Simulation complete." << std::endl;
	}
}