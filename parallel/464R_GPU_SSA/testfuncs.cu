﻿#include "testfuncs.cuh"

__host__  void test_updateSimsAndPropensities() {
	const int s = 3;
	const int n = 3;
	const int m = 3;
	const int max_reactants = 2;
	const int max_affected = 3;
	double propensities[s * m] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	double reaction_rates[n] = { 1, 2, 0.5 };
	// 2A -> B, 1
	// A 2B -> C, 2
	// C -> 2C, 0.5
	int reactants[n * max_reactants * 2] = { 0, 2, 0, 0, 0, 1, 1, 2, 2, 1, 0, 0 };
	int state_changes[m * n] = { -2, 1, 0, -1, -2, 1, 0, 0, 1 };
	int dep_matrix[n * max_affected] = { 0, 1, -1, 0, 1, 2, 2, -1, -1 };
	int sim_configs[m * s] = { 5, 5, 5, 5, 5, 5, 5, 5, 5 };
	int fired_reactions[s] = { 0, 1, 2 };
	bool stability_flags[s] = { false, false, false };

	double* dev_props = 0;
	double* dev_rates = 0;
	int* dev_reactants = 0;
	int* dev_state_changes = 0;
	int* dev_dep_matrix = 0;
	int* dev_fired_reactions = 0;
	int* dev_sim_configs = 0;
	bool* dev_stability_flags = 0;

	hipMalloc(&dev_props, s * m * sizeof(double));
	hipMalloc(&dev_rates, m * sizeof(double));
	hipMalloc(&dev_reactants, n * max_reactants * 2 * sizeof(int));
	hipMalloc(&dev_state_changes, m * n * sizeof(int));
	hipMalloc(&dev_dep_matrix, n * max_affected * sizeof(int));
	hipMalloc(&dev_sim_configs, m * s * sizeof(int));
	hipMalloc(&dev_fired_reactions, s * sizeof(int));
	hipMalloc(&dev_stability_flags, s * sizeof(bool));

	hipMemcpy(dev_props, propensities, s * m * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_rates, reaction_rates, m * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_reactants, reactants, n * max_reactants * 2 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_state_changes, state_changes, m * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_dep_matrix, dep_matrix, n * max_affected * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_sim_configs, sim_configs, m * s * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_fired_reactions, fired_reactions, s * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_stability_flags, stability_flags, s * sizeof(bool), hipMemcpyHostToDevice);

	dim3 grid(1, 1);
	dim3 threads(16, 16);
	//updateSims(s, n, dev_fired_reactions, dev_sim_configs, dev_state_changes, dev_stability_flags);
	//updateProps(s, n, m, max_reactants, dev_sim_configs, dev_reactants, dev_rates, dev_props);

	hipMemcpy(propensities, dev_props, s * m * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(reaction_rates, dev_rates, m * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(reactants, dev_reactants, n * max_reactants * 2 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(state_changes, dev_state_changes, m * n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(dep_matrix, dev_dep_matrix, n * max_affected * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(sim_configs, dev_sim_configs, m * s * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(fired_reactions, dev_fired_reactions, s * sizeof(int), hipMemcpyDeviceToHost);

	std::cout << "Simulation Configs" << '\n';
	for (int i = 0; i < s * n; i++) {
		std::cout << sim_configs[i] << " ";
	}
	std::cout << "\n\n";

	std::cout << "Propensities" << '\n';
	for (int i = 0; i < s * m; i++) {
		std::cout << propensities[i] << " ";
	}
}

__host__ bool test_scaleRVs(int s, int m, bool verbose) {
	thrust::host_vector<double> propensities(s * m);
	for (int i = 0; i < s; i++) {
		for (int j = 0; j < m; j++) {
			propensities[i * m + j] = j + 1;
		}
	}

	// Construct keys for row-wise inclusive scan
	thrust::host_vector<int> keys(s * m);
	for (int i = 0; i < s; i++) {
		for (int j = 0; j < m; j++) {
			keys[i * m + j] = i; // key for each element in a row is just its row number
		}
	}

	thrust::host_vector<double> propscan(s * m);
	thrust::inclusive_scan_by_key(thrust::host, keys.begin(), keys.end(), propensities.begin(), propscan.begin());

	std::default_random_engine generator;
	std::uniform_real_distribution<double> distribution(0.0, 1.0);

	thrust::host_vector<double> uniformRVs(s);
	for (int i = 0; i < s; i++) {
		uniformRVs[i] = distribution(generator);
	}

	thrust::host_vector<double> scaledRVs(s);
	for (int i = 0; i < s; i++) {
		scaledRVs[i] = uniformRVs[i] * propscan[m + i * m - 1];
	}

	thrust::device_vector<double> dev_propscan = propscan;
	thrust::device_vector<double> dev_uniformRVs = uniformRVs;
	//scaleRVs(thrust::raw_pointer_cast(dev_uniformRVs.data()), thrust::raw_pointer_cast(dev_propscan.data()), s, m);
	
	thrust::host_vector<double> par_scaledRVs = dev_uniformRVs;

	int fail_index = -1;
	for (int i = 0; i < s; i++) {
		if (par_scaledRVs[i] != scaledRVs[i]) {
			fail_index = i;
			break;
		}
	}

	if (fail_index != -1) {
		std::cout << "Failure for s = " << s << " and m = " << m << '\n';
		std::cout << "Fails first at index i = " << fail_index << "\n\n";
		if (verbose) {
			std::cout << "Random Variables" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << uniformRVs[i] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Propensity Sums" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << propscan[m + i * m - 1] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Host Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << scaledRVs[i] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Device Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << par_scaledRVs[i] << ", ";
			}
			std::cout << "}\n\n";
		}
		return false;
	}
	else {
		std::cout << "Success for s = " << s << " and m = " << m << "\n\n";
		if (verbose) {
			std::cout << "Host Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << scaledRVs[i] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Device Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << par_scaledRVs[i] << ", ";
			}
			std::cout << "}\n\n";
		}
		return true;
	}
}

__host__ bool test_checkBins(int s, int m, bool verbose) {
	thrust::host_vector<double> propensities(s * m);
	for (int i = 0; i < s; i++) {
		for (int j = 0; j < m; j++) {
			propensities[i * m + j] = j + 1;
		}
	}

	// Construct keys for row-wise inclusive scan
	thrust::host_vector<int> keys(s * m);
	for (int i = 0; i < s; i++) {
		for (int j = 0; j < m; j++) {
			keys[i * m + j] = i; // key for each element in a row is just its row number
		}
	}

	thrust::host_vector<double> propscan(s * m);
	thrust::inclusive_scan_by_key(thrust::host, keys.begin(), keys.end(), propensities.begin(), propscan.begin());

	std::default_random_engine generator;
	std::uniform_real_distribution<double> distribution(0.0, 1.0);

	thrust::host_vector<double> uniformRVs(s);
	for (int i = 0; i < s; i++) {
		uniformRVs[i] = distribution(generator);
	}

	thrust::device_vector<double> dev_propscan = propscan;
	thrust::device_vector<double> dev_uniformRVs = uniformRVs;
	//scaleRVs(thrust::raw_pointer_cast(dev_uniformRVs.data()), thrust::raw_pointer_cast(dev_propscan.data()), s, m);

	// move scaled RVs back to host
	uniformRVs = dev_uniformRVs;

	thrust::host_vector<int> bins(s);
	thrust::device_vector<int> dev_bins(s);
	thrust::fill(dev_bins.begin(), dev_bins.end(), -1);

	// Compute bins on host
	for (int i = 0; i < s; i++) {
		double rv = uniformRVs[i];
		for (int j = 0; j < m; j++) {
			double left_edge, right_edge;
			if (j == 0) {
				left_edge = 0;
				right_edge = propscan[i * m + j];
			}
			else {
				left_edge = propscan[i * m + j - 1];
				right_edge = propscan[i * m + j];
			}

			// Push right edge over to include total for last element
			if (j == m - 1) {
				right_edge += 1;
			}

			if (left_edge <= rv && rv < right_edge) {
				bins[i] = j;
			};
		}
	}

	//checkBins(thrust::raw_pointer_cast(dev_propscan.data()), thrust::raw_pointer_cast(dev_uniformRVs.data()), thrust::raw_pointer_cast(dev_bins.data()), s, m);

	thrust::host_vector<int> par_bins = dev_bins;

	int fail_index = -1;
	for (int i = 0; i < s; i++) {
		if (par_bins[i] != bins[i]) {
			fail_index = i;
			break;
		}
	}

	if (fail_index != -1) {
		std::cout << "Failure for s = " << s << " and m = " << m << '\n';
		std::cout << "Fails first at index i = " << fail_index << "\n\n";
		if (verbose) {
			std::cout << "Random Variables" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << uniformRVs[i] << ", ";
			}
			std::cout << "}\n\n\n";

			std::cout << "Propensity Sums" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << propscan[m + i * m - 1] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Host Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << bins[i] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Device Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << par_bins[i] << ", ";
			}
			std::cout << "}\n\n";
		}
		return false;
	}
	else {
		std::cout << "Success for s = " << s << " and m = " << m << "\n\n";
		if (verbose) {
			std::cout << "Host Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << bins[i] << ", ";
			}
			std::cout << "}\n\n";

			std::cout << "Device Result" << '\n';
			std::cout << "{";
			for (int i = 0; i < s; i++) {
				std::cout << par_bins[i] << ", ";
			}
			std::cout << "}\n\n";
		}
		return true;
	}
}