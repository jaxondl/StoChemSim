#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"

// TESETER: Tarek
// Incremement times by drawn RVs
// double[] randomVariables: The random variables array on device
// double[] times: The array of times on device
// size_t s: the number of simulations
__global__ void updateTimesKernel(double* randomVariables, double* times, size_t s) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < s) {
		times[gid] += randomVariables[gid];
	}
}

// TESTER: Tarek
// Multiply the elements of 1D array x1 by the last column of 2D array x2 in-place
// double[] x1: 1D Array which is multiplied and stores the output
// double[] x2: 2D Array which multiplies x1 in place
// int n: Offset for each row in order to grab the final entry
// int size: overall size of x1 and the number of rows in x2
__global__ void offsetMultiplicationKernel(double* x1, double* x2, size_t n, size_t size) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	// Ignore excess threads for a multi-block invocation
	if (gid < size) {
		x1[gid] *= x2[n + gid * n - 1];
	}
}

// TESTER: Vidur
// fired_reactions: list (length s) of the IDs of the reaction fired for each simulation
// sim_configs: matrix giving configuration for each simulation
// state_changes: State change vector for each reaction
__global__ void updateSimsKernel(int s, int n, int* fired_reactions, int* sim_configs, int* state_changes, bool* stability_flags) {
	// Add reactions vectors given by checkBins() to simulation configs
	int gid = blockIdx.y * n + blockIdx.x * blockDim.x + threadIdx.x;
	int rid = blockIdx.x * blockDim.x + threadIdx.x;

	// Only progress if within bounds and simulation has not reached stability
	if (rid < n && stability_flags[blockIdx.y] == false) {
		sim_configs[gid] = sim_configs[gid] + state_changes[(fired_reactions[blockIdx.y]) * n + rid];
	}
}

__global__ void updatePropsKernel(int s, int n, int m, int max_reactants, int* sim_configs, int* reactants, double* reaction_rates, double* propensities) {
	// Each block updates one propensity value
	int gid = blockIdx.y * m + blockIdx.x * blockDim.x + threadIdx.x;
	int rid = blockIdx.x * blockDim.x + threadIdx.x;

	if (rid < m) {
		int reaction_num = rid; // gid % m;
		int reactants_starting_idx = reaction_num * max_reactants * 2;
		double propensity = reaction_rates[reaction_num];
		for (int i = 0; i < max_reactants; i++) {
			int molecule_idx = i * 2 + reactants_starting_idx;
			int reactant_coef_idx = i * 2 + reactants_starting_idx + 1;
			int molecule_amt = sim_configs[blockIdx.y * n + reactants[molecule_idx]]; // [sim*n + reactants[molecule_idx]];
			for (int j = 0; j < reactants[reactant_coef_idx]; j++) {
				propensity *= (molecule_amt - j);
			}
		}
		propensities[gid] = propensity;
	}
}

// TESTER: Zhecheng
// helper function for exponential RVs
__global__ void calculateExponentialRVsKernel(double* randomVariables, double* propscan, size_t s, size_t m) {
	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (gid < s) {
		randomVariables[gid] = -1 / propscan[m + gid * m - 1] * log(1 - randomVariables[gid]); // replace each uniform variable in first half of RV array with exponential counterpart
	}
}

// TESTER: Tarek
// Each row of blocks corresponds to a single simulation (gridDim.y = s)
// Each row has enough blocks to create enough threads to check every bin location at once (gridDim.x * blockDim.x >= m)
__global__ void checkBinsKernel(double* propensity_scan, double* uniformRVs, int* bins, size_t s, size_t m) {
	int tid = threadIdx.x;
	int rid = blockIdx.x * blockDim.x + tid;
	int gid = blockIdx.y * m + rid;

	// only threads with row index less than the number of bins will check so that excess threads in last block are excluded
	if (rid < m) {
		// Move uniform RV for simulation and the bins being checked to shared memory
		double urv = uniformRVs[blockIdx.y]; // make sure uniformRVs is pointer offset

		// TODO: Would be made more efficient by padding with 0 at the beginning of the scan, as this would eliminate the if statement
		// TODO: Each value is loaded twice, should be changed to avoid this
		double left_edge, right_edge;
		if (rid == 0) {
			left_edge = 0;
			right_edge = propensity_scan[gid];
		}
		else {
			left_edge = propensity_scan[gid - 1];
			right_edge = propensity_scan[gid];
		}

		// Last thread in row increases its right edge to include 1 in the boundary
		if (rid == m - 1) {
			right_edge += 1;
		}

		// Only one warp per row will diverge on this instruction
		if (left_edge <= urv && urv < right_edge) {
			bins[blockIdx.y] = rid;
		}
	}
}

__global__ void propCheckKernel(double* propscan, bool* stability_flags, int s, int m) {
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + tid;
	double prop_row_sum = propscan[m + tid * m - 1];

	// only set stability to true if the thread is within bounds and the simulation propensity sum is nonzero
	if (gid < s && prop_row_sum == 0) {
		stability_flags[gid] = true;
	}
}